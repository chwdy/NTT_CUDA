#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>

#include "../include/utils2.h"	
#include "../include/utils.h"
/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.cuh" //INCLUDE HEADER FILE
#include "../include/utils_device.cuh"	
#include "../include/cuda_device.cuh"

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
using namespace std;


__global__ void ntt_cuda_kernel_stepA(uint64_t *g_idata, uint64_t num_bits, uint64_t n, uint64_t p, uint64_t r, bool rev, uint64_t *g_odata)
{
	uint64_t m, k_, a, factor1, factor2;
	//set thread ID
	uint64_t tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	//boundary check
	if (tid >= n || idx >n)return;
	if (rev)
	{
		uint64_t reverse_num= 0;
		for(uint64_t j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(idx & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		g_odata[reverse_num] = g_idata[idx];
	}
	else
	{
		g_odata[idx] = g_idata[idx];
	}
	__syncthreads();
	if (idx == 0)
	{
		for (uint64_t i = 1; i <= log2_D(n); i++)
		{
			m = pow_D(uint64_t(2), i);
			k_ = (p - 1) / m;
			a = modExp_D(r, k_, p);
			for (uint64_t j = 0; j < n; j += m)
			{
				for (uint64_t k = 0; k < m / 2; k++)
				{
					factor1 = g_odata[j + k];
					factor2 = modulo_D(modExp_D(a, k, p) * g_odata[j + k + m / 2], p);
					g_odata[j + k] = modulo_D(factor1 + factor2, p);
					g_odata[j + k + m / 2] = modulo_D(factor1 - factor2, p);
				}
			}
		}
	}

}

extern "C" 
uint64_t *inPlaceNTT_DIT_stepA(uint64_t *vec, uint64_t n, uint64_t p, uint64_t r, bool rev)
{

	double computestart, computeElaps,copystart,copyElaps;

	int blocksize = 1024;
	dim3 block(blocksize, 1);
	dim3 grid((n - 1) / block.x + 1, 1);

	//var init
	size_t bytes = n * sizeof(uint64_t);
	uint64_t *vec_host = (uint64_t *)malloc(bytes);
	uint64_t *outVec_host = (uint64_t *)malloc(bytes); //grid.x * sizeof(uint64_t));
	//printf("grid %d block %d \n", grid.x, block.x);

	memcpy(vec_host, vec, bytes);

	// device memory declare
	uint64_t *vec_dev = NULL;
	uint64_t *outVec_dev = NULL;

	//device memory allocate
	CHECK(hipMalloc((void **)&vec_dev, bytes));
	CHECK(hipMalloc((void **)&outVec_dev, bytes));


	//remove bitreversal
	uint64_t num_bits = log2(n);
	CHECK(hipMemset(vec_dev,0,bytes))
	CHECK(hipMemset(outVec_dev,0,bytes))
	copystart= cpuSecond();
	CHECK(hipMemcpy(vec_dev, vec_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	computestart= cpuSecond();
	ntt_cuda_kernel_stepA<<<grid, block>>>(vec_dev,num_bits, n, p, r, rev, outVec_dev);
	CHECK(hipDeviceSynchronize());	
	computeElaps = 1000 * (cpuSecond() - computestart);
	CHECK(hipMemcpy(outVec_host, outVec_dev, bytes, hipMemcpyDeviceToHost));
	copyElaps = 1000 * (cpuSecond() - copystart);
	printf("gpu 1 pure compute time: %lf compute+copy time: %lf for ### bit reversal### \n first two number is %lld %lld \n", computeElaps, copyElaps,outVec_host[0],outVec_host[1]);


	CHECK(hipFree(vec_dev));
	CHECK(hipFree(outVec_dev));

	return outVec_host;
}
