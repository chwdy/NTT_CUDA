#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>

#include "../include/utils2.h"	
#include "../include/utils.h"
/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.cuh" //INCLUDE HEADER FILE
#include "../include/utils_device.cuh"	
#include "../include/cuda_device.cuh"

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
using namespace std;

__global__ void ntt_cuda_kernel_stepB2_rev(uint64_t *g_idata, int num_bits ,uint64_t *n,  bool rev, uint64_t *g_odata)
{
	//set thread ID
	uint64_t tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	//boundary check
	if (tid >= *n || idx >*n)return;
	if (rev)
	{
		uint64_t reverse_num= 0;
		for(uint64_t j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(idx & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		g_odata[reverse_num] = g_idata[idx];
	}
	else
	{
		g_odata[idx] = g_idata[idx];
	}
}
__global__ void ntt_cuda_kernel_stepB2_fac_A(uint64_t *g_idata, uint64_t *table ,uint64_t *n, uint64_t *p, uint64_t i,uint64_t *fac1_dev,uint64_t *fac2_dev,uint64_t *g_odata)
{
	//set thread ID
	uint64_t tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	//boundary check
	if (tid >= *n || idx >*n)return;
	uint64_t m = pow_D(uint64_t(2), i);
	uint64_t k = idx %m;

	if(k<m/2){
		fac1_dev[idx] = g_odata[idx];
		fac2_dev[idx] = modulo_D(uint64_t(table[(i-1)*2048+k])*uint64_t(g_odata[idx+ m / 2]), *p);
	}else{
		fac1_dev[idx] = g_odata[idx-m/2];
		fac2_dev[idx] = modulo_D(uint64_t(table[(i-1)*2048+k-m/2])*uint64_t(g_odata[idx]), *p);
	}	
}
__global__ void ntt_cuda_kernel_stepB2_fac_B(uint64_t *g_idata,uint64_t *table ,uint64_t *n, uint64_t *p, uint64_t i, uint64_t *fac1_dev,uint64_t *fac2_dev,uint64_t *g_odata)
{
	//set thread ID
	uint64_t tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	//boundary check
	if (tid >= *n || idx >= *n)return;
	uint64_t m = pow_D(uint64_t(2), i);
	uint64_t k = idx %m;
	if(k<m/2){
		g_odata[idx] = modulo_D(fac1_dev[idx] + fac2_dev[idx], *p);
	}else{
		g_odata[idx] = modulo_D(fac1_dev[idx] - fac2_dev[idx], *p);
	}
}

extern "C" 
uint64_t *inPlaceNTT_DIT_stepB2(uint64_t *vec, uint64_t n, uint64_t p, uint64_t r, bool rev)
{
	double computestart, computeElaps,copystart,copyElaps;

	int blocksize = 1024;
	dim3 block(blocksize, 1);
	dim3 grid((n - 1) / block.x + 1, 1);

	//var init
	size_t bytes = n * sizeof(uint64_t);
	uint64_t *vec_host = (uint64_t *)malloc(bytes);
	uint64_t *outVec_host = (uint64_t *)malloc(bytes);

	memcpy(vec_host, vec, bytes);

	// device memory declare
	uint64_t *vec_dev = NULL;
	uint64_t *outVec_dev = NULL;

	//device memory allocate
	CHECK(hipMalloc((void **)&vec_dev, bytes));
	CHECK(hipMalloc((void **)&outVec_dev, bytes));

	//remove bitreversal
	uint64_t num_bits = log2(n);

	//modexp offline
	num_bits = log2(n);
	uint64_t a_table [32];
	int i,j;
	for (i=1;i<=32;i++){
		a_table[i-1] = modExp(r,(p-1)/pow(2,i),p);
	}
	uint64_t ak_table [65536] ;
	for (i=0;i<32;i++){
		for (j=0;j<2048;j++){
		ak_table[i*2048+j] = modExp(a_table[i],j,p);
		}
	}
	uint64_t *ak_table_dev =NULL;
	uint64_t *n_dev =NULL;
	uint64_t *p_dev =NULL;
	uint64_t *fac1_dev =NULL;
	uint64_t *fac2_dev =NULL;

	CHECK(hipMalloc((void **)&ak_table_dev, sizeof(ak_table)));
	CHECK(hipMalloc((void **)&n_dev, sizeof(n)));
	CHECK(hipMalloc((void **)&p_dev, sizeof(p)));
	CHECK(hipMalloc((void **)&fac1_dev, bytes));
	CHECK(hipMalloc((void **)&fac2_dev, bytes));
	copystart= cpuSecond();
	CHECK(hipMemcpy(ak_table_dev, ak_table, sizeof(ak_table), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(n_dev, &n, sizeof(n), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(p_dev, &p, sizeof(p), hipMemcpyHostToDevice));

	CHECK(hipMemset(vec_dev,0,bytes))
	CHECK(hipMemset(outVec_dev,0,bytes))
	
	CHECK(hipMemcpy(vec_dev, vec_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	computestart= cpuSecond();
	ntt_cuda_kernel_stepB2_rev<<<grid, block>>>(vec_dev,num_bits,n_dev,rev, outVec_dev);
	CHECK(hipDeviceSynchronize());	
	for (uint64_t i = 1; i <= num_bits; i++)
	{
		ntt_cuda_kernel_stepB2_fac_A<<<grid, block>>>(vec_dev,ak_table_dev,n_dev,p_dev, i,fac1_dev,fac2_dev,outVec_dev);
		CHECK(hipDeviceSynchronize());	
		ntt_cuda_kernel_stepB2_fac_B<<<grid, block>>>(vec_dev,ak_table_dev,n_dev,p_dev, i,fac1_dev,fac2_dev,outVec_dev);
		CHECK(hipDeviceSynchronize());	
	}
	computeElaps = 1000 * (cpuSecond() - computestart);
	CHECK(hipMemcpy(outVec_host, outVec_dev, bytes, hipMemcpyDeviceToHost));
	copyElaps = 1000 * (cpuSecond() - copystart);
	printf("gpu 1 pure compute time: %lf compute+copy time: %lf for ### modexp offline### \n first two number is %lld %lld \n", computeElaps, copyElaps,outVec_host[0],outVec_host[1]);

	CHECK(hipFree(vec_dev));
	CHECK(hipFree(ak_table_dev));
	CHECK(hipFree(n_dev));
	CHECK(hipFree(p_dev));
	CHECK(hipFree(fac1_dev));
	CHECK(hipFree(fac2_dev));
	CHECK(hipFree(outVec_dev));

	return outVec_host;
}
