#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>

#include "../include/utils2.h"	
#include "../include/utils.h"
/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.cuh" //INCLUDE HEADER FILE
#include "../include/utils_device.cuh"	
#include "../include/cuda_device.cuh"

/** Pure single thread implementation
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
using namespace std;


__global__ void ntt_kernel_cuda_asis(uint64_t *g_idata, uint64_t n, uint64_t p, uint64_t r, bool rev, uint64_t *g_odata)
{

	uint64_t m, k_, a, factor1, factor2;
	//set thread ID
	uint64_t tid = threadIdx.x;
	//boundary check
	if (tid >= n)return;
	if (rev)
	{
		if (blockIdx.x == 0 && tid == 0)
		{
			uint64_t *temp;
			temp = (uint64_t *) malloc(n*sizeof(uint64_t));
			temp = bit_reverse_D(g_idata, n);
			memcpy(g_odata, temp, n * sizeof(uint64_t));
		}
	}
	else
	{
		for (uint64_t i = 0; i < n; i++)
		{
			g_odata[i] = g_idata[i];
		}
	}
	
	if (blockIdx.x == 0 && tid == 0)
	{
		//printf("godata first two number is %d %d \n",  g_odata[55],g_odata[71]);
		for (uint64_t i = 1; i <= log2_D(n); i++)
		{
			m = pow_D(uint64_t(2), i);
			k_ = (p - 1) / m;
			a = modExp_D(r, k_, p);
			for (uint64_t j = 0; j < n; j += m)
			{
				for (uint64_t k = 0; k < m / 2; k++)
				{
					factor1 = g_odata[j + k];
					factor2 = modulo_D(modExp_D(a, k, p) * g_odata[j + k + m / 2], p);
					g_odata[j + k] = modulo_D(factor1 + factor2, p);
					g_odata[j + k + m / 2] = modulo_D(factor1 - factor2, p);
				}
			}
		}
	}
	__syncthreads();
}

extern "C" 
uint64_t *inPlaceNTT_DIT_cuda_asis(uint64_t *vec, uint64_t n, uint64_t p, uint64_t r, bool rev)
{

	double computestart, computeElaps,copystart,copyElaps;

	int blocksize = 1024;
	dim3 block(blocksize, 1);
	dim3 grid((n - 1) / block.x + 1, 1);

	//var init
	size_t bytes = n * sizeof(uint64_t);
	uint64_t *vec_host = (uint64_t *)malloc(bytes);
	uint64_t *outVec_host = (uint64_t *)malloc(bytes); //

	memcpy(vec_host, vec, bytes);

	// device memory declare
	uint64_t *vec_dev = NULL;
	uint64_t *outVec_dev = NULL;

	//device memory allocate
	CHECK(hipMalloc((void **)&vec_dev, bytes));
	CHECK(hipMalloc((void **)&outVec_dev, bytes));

	copystart= cpuSecond();
	//first task
	CHECK(hipMemset(vec_dev,0,bytes))
	CHECK(hipMemset(outVec_dev,0,bytes))
	
	CHECK(hipMemcpy(vec_dev, vec_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	computestart= cpuSecond();
	ntt_kernel_cuda_asis<<<grid, block>>>(vec_dev, n, p, r, rev, outVec_dev);
	CHECK(hipDeviceSynchronize());
	computeElaps = 1000 * (cpuSecond() - computestart);
	CHECK(hipMemcpy(outVec_host, outVec_dev, bytes, hipMemcpyDeviceToHost));
	copyElaps = 1000 * (cpuSecond() - copystart);
	printf("gpu 1 pure compute time: %lf compute+copy time: %lf for ### first task ### \n first two number is %lld %lld \n", computeElaps, copyElaps,outVec_host[0],outVec_host[1]);


	CHECK(hipFree(vec_dev));
	CHECK(hipFree(outVec_dev));


	return outVec_host;
}
