#include "hip/hip_runtime.h"
#include <cstdint> /* uint64_t */
#include <cstdlib> /* malloc() */
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h> /* log2(), pow() */

#include "../include/cuda_device.cuh"
#include "../include/utils.h"
#include "../include/utils2.h"
#include "../include/utils_device.cuh"
/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.cuh" //INCLUDE HEADER FILE

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey
 * NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
using namespace std;

__global__ void ntt_cuda_kernel_rev(uint64_t *g_idata, int offset, int num_bits,
                                    uint64_t *n, bool rev, uint64_t *g_odata) {
  // uint64_t m, factor1, factor2;
  // set thread ID
  uint64_t tid = threadIdx.x;
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  // boundary check
  if (tid >= *n || idx > *n)
    return;
  if (rev) {
    uint64_t reverse_num = 0;
    for (uint64_t j = 0; j < num_bits; j++) {
      reverse_num = reverse_num << 1;
      if (idx & (1 << j)) {
        reverse_num = reverse_num | 1;
      }
    }
    g_odata[offset * *n + reverse_num] = g_idata[offset * *n + idx];
  } else {
    g_odata[offset * *n + idx] = g_idata[offset * *n + idx];
  }
}
__global__ void ntt_cuda_kernel_fac_A(uint64_t *g_idata, int offset,
                                      uint64_t *table, uint64_t *n, uint64_t *p,
                                      uint64_t i, uint64_t *fac1_dev,
                                      uint64_t *fac2_dev, uint64_t *g_odata) {
  // set thread ID
  uint64_t tid = threadIdx.x;
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  // boundary check
  if (tid >= *n || idx > *n)
    return;
  uint64_t m = pow_D(uint64_t(2), i);
  uint64_t k = idx % m;

  if (k < m / 2) {
    fac1_dev[offset * *n + idx] = g_odata[offset * *n + idx];
    fac2_dev[offset * *n + idx] =
        modulo_D(uint64_t(table[(i - 1) * 2048 + k]) *
                     uint64_t(g_odata[offset * *n + idx + m / 2]),
                 *p);
  } else {
    fac1_dev[offset * *n + idx] = g_odata[offset * *n + idx - m / 2];
    fac2_dev[offset * *n + idx] =
        modulo_D(uint64_t(table[(i - 1) * 2048 + k - m / 2]) *
                     uint64_t(g_odata[offset * *n + idx]),
                 *p);
  }
}
__global__ void ntt_cuda_kernel_fac_B(uint64_t *g_idata, int offset,
                                      uint64_t *table, uint64_t *n, uint64_t *p,
                                      uint64_t i, uint64_t *fac1_dev,
                                      uint64_t *fac2_dev, uint64_t *g_odata)

{
  // uint64_t factor1, factor2;
  // set thread ID
  uint64_t tid = threadIdx.x;
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  // boundary check
  if (tid >= *n || idx > *n)
    return;
  uint64_t m = pow_D(uint64_t(2), i);
  uint64_t k = idx % m;
  if (k < m / 2) {
    g_odata[offset * *n + idx] =
        modulo_D(fac1_dev[offset * *n + idx] + fac2_dev[offset * *n + idx], *p);
  } else {
    g_odata[offset * *n + idx] =
        modulo_D(fac1_dev[offset * *n + idx] - fac2_dev[offset * *n + idx], *p);
  }
}

extern "C" uint64_t *inPlaceNTT_DIT(uint64_t **vec, uint64_t batch_size,
                                    uint64_t n, uint64_t p, uint64_t r,
                                    bool rev) {


  int blocksize = 1024;
  dim3 block(blocksize, 1);
  dim3 grid((n - 1) / block.x + 1, 1);

  // var init
  size_t bytes = n * batch_size * sizeof(uint64_t);
  uint64_t *vec_host = (uint64_t *)malloc(bytes);
  uint64_t *outVec_host =
      (uint64_t *)malloc(bytes); // grid.x * sizeof(uint64_t));

  for (int i = 0; i < batch_size; i++) {

    memcpy(&vec_host[i * n], vec[i], n * sizeof(uint64_t));
  }

  // device memory declare
  uint64_t *vec_dev = NULL;
  uint64_t *outVec_dev = NULL;

  // device memory allocate
  CHECK(hipMalloc((void **)&vec_dev, bytes));
  CHECK(hipMalloc((void **)&outVec_dev, bytes));

  // remove bitreversal
  uint64_t num_bits = log2(n);

  num_bits = log2(n);
  // pre-computed-modEXP
  uint64_t a_table[32];
  int i, j;
  for (i = 1; i <= 32; i++) {
    a_table[i - 1] = modExp(r, (p - 1) / pow(2, i), p);
  }
  uint64_t ak_table[65536];
  for (i = 0; i < 32; i++) {
    for (j = 0; j < 2048; j++) {
      ak_table[i * 2048 + j] = modExp(a_table[i], j, p);
    }
  }
  uint64_t *ak_table_dev = NULL;
  uint64_t *n_dev = NULL;
  uint64_t *p_dev = NULL;
  uint64_t *fac1_dev = NULL;
  uint64_t *fac2_dev = NULL;

  CHECK(hipMalloc((void **)&ak_table_dev, sizeof(ak_table)));
  CHECK(hipMalloc((void **)&n_dev, sizeof(n)));
  CHECK(hipMalloc((void **)&p_dev, sizeof(p)));
  CHECK(hipMalloc((void **)&fac1_dev, bytes));
  CHECK(hipMalloc((void **)&fac2_dev, bytes));
  CHECK(hipMemcpy(ak_table_dev, ak_table, sizeof(ak_table),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(n_dev, &n, sizeof(n), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(p_dev, &p, sizeof(p), hipMemcpyHostToDevice));

  CHECK(hipMemset(vec_dev, 0, bytes))
  CHECK(hipMemset(outVec_dev, 0, bytes))

  CHECK(hipMemcpy(vec_dev, vec_host, bytes, hipMemcpyHostToDevice));
  CHECK(hipDeviceSynchronize());

  for (int offset = 0; offset < batch_size; offset++) {

    ntt_cuda_kernel_rev<<<grid, block>>>(vec_dev, offset, num_bits, n_dev, rev,
                                         outVec_dev);
    CHECK(hipDeviceSynchronize());
    for (uint64_t i = 1; i <= num_bits; i++) {
      ntt_cuda_kernel_fac_A<<<grid, block>>>(vec_dev, offset, ak_table_dev,
                                             n_dev, p_dev, i, fac1_dev,
                                             fac2_dev, outVec_dev);
      CHECK(hipDeviceSynchronize());
      ntt_cuda_kernel_fac_B<<<grid, block>>>(vec_dev, offset, ak_table_dev,
                                             n_dev, p_dev, i, fac1_dev,
                                             fac2_dev, outVec_dev);
      CHECK(hipDeviceSynchronize());
    }
  }
  CHECK(hipDeviceSynchronize());
  CHECK(hipMemcpy(outVec_host, outVec_dev, bytes, hipMemcpyDeviceToHost));

  CHECK(hipFree(ak_table_dev));
  CHECK(hipFree(n_dev));
  CHECK(hipFree(p_dev));
  CHECK(hipFree(vec_dev));
  CHECK(hipFree(outVec_dev));

  return outVec_host;
}
