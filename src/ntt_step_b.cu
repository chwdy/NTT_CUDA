#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>

#include "../include/utils2.h"	
#include "../include/utils.h"
/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.cuh" //INCLUDE HEADER FILE
#include "../include/utils_device.cuh"	
#include "../include/cuda_device.cuh"

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
using namespace std;

__global__ void ntt_cuda_kernel_stepB(uint64_t *g_idata, int num_bits,uint64_t *table ,uint64_t *n, uint64_t *p, bool rev, uint64_t *g_odata)
{

	uint64_t m, factor1, factor2;
	//set thread ID
	uint64_t tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	//boundary check
	if (tid >= *n || idx >*n)return;
	if (rev)
	{
		uint64_t reverse_num= 0;
		for(uint64_t j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(idx & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		g_odata[reverse_num] = g_idata[idx];
	}
	else
	{
		g_odata[idx] = g_idata[idx];
	}
	__syncthreads();
	if (idx == 0)
	{
		for (uint64_t i = 1; i <= num_bits; i++)
		{
			m = pow_D(uint64_t(2), i);
			for (uint64_t j = 0; j < *n; j += m)
			{
				for (uint64_t k = 0; k < m / 2; k++)
				{
					factor1 = g_odata[j + k];
					factor2 = modulo_D(uint64_t(table[(i-1)*2048+k])*uint64_t(g_odata[j + k + m / 2]), *p);
					g_odata[j + k] = modulo_D(factor1 + factor2, *p);
					g_odata[j + k + m / 2] = modulo_D(factor1 - factor2, *p);
				}
			}
		}	
		
	}

}
extern "C" 
uint64_t *inPlaceNTT_DIT_stepB(uint64_t *vec, uint64_t n, uint64_t p, uint64_t r, bool rev)
{

	double computestart, computeElaps,copystart,copyElaps;

	int blocksize = 1024;
	dim3 block(blocksize, 1);
	dim3 grid((n - 1) / block.x + 1, 1);

	//var init
	size_t bytes = n * sizeof(uint64_t);
	uint64_t *vec_host = (uint64_t *)malloc(bytes);
	uint64_t *outVec_host = (uint64_t *)malloc(bytes); //grid.x * sizeof(uint64_t));
	//printf("grid %d block %d \n", grid.x, block.x);

	memcpy(vec_host, vec, bytes);

	// device memory declare
	uint64_t *vec_dev = NULL;
	uint64_t *outVec_dev = NULL;

	//device memory allocate
	CHECK(hipMalloc((void **)&vec_dev, bytes));
	CHECK(hipMalloc((void **)&outVec_dev, bytes));

	//remove bitreversal
	uint64_t num_bits = log2(n);

	//modexp offline
	num_bits = log2(n);
	uint64_t a_table [32];
	int i,j;
	for (i=1;i<=32;i++){
		a_table[i-1] = modExp(r,(p-1)/pow(2,i),p);
		//printf("A: %llu i: %d \n",a_table[i-1],i);
	}
	uint64_t ak_table [65536] ;
	for (i=0;i<32;i++){
		for (j=0;j<2048;j++){
		ak_table[i*2048+j] = modExp(a_table[i],j,p);
		}
	}
	uint64_t *ak_table_dev =NULL;
	uint64_t *n_dev =NULL;
	uint64_t *p_dev =NULL;
	//uint64_t *r_dev =NULL;

	CHECK(hipMalloc((void **)&ak_table_dev, sizeof(ak_table)));
	CHECK(hipMalloc((void **)&n_dev, sizeof(n)));
	CHECK(hipMalloc((void **)&p_dev, sizeof(p)));
	//CHECK(hipMalloc((void **)&r_dev, sizeof(r)));
	CHECK(hipMemcpy(ak_table_dev, ak_table, sizeof(ak_table), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(n_dev, &n, sizeof(n), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(p_dev, &p, sizeof(p), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(r_dev, &r, sizeof(r), hipMemcpyHostToDevice));

	CHECK(hipMemset(vec_dev,0,bytes))
	CHECK(hipMemset(outVec_dev,0,bytes))
	copystart= cpuSecond();
	CHECK(hipMemcpy(vec_dev, vec_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	computestart= cpuSecond();
	ntt_cuda_kernel_stepB<<<grid, block>>>(vec_dev,num_bits,ak_table_dev,n_dev, p_dev,rev, outVec_dev);
	CHECK(hipDeviceSynchronize());	
	computeElaps = 1000 * (cpuSecond() - computestart);
	CHECK(hipMemcpy(outVec_host, outVec_dev, bytes, hipMemcpyDeviceToHost));
	copyElaps = 1000 * (cpuSecond() - copystart);
	printf("gpu 1 pure compute time: %lf compute+copy time: %lf for ### modexp offline### \n first two number is %lld %lld \n", computeElaps, copyElaps,outVec_host[0],outVec_host[1]);

	CHECK(hipFree(ak_table_dev));
	CHECK(hipFree(n_dev));
	CHECK(hipFree(p_dev));
	//CHECK(hipFree(r_dev));
	CHECK(hipFree(vec_dev));
	CHECK(hipFree(outVec_dev));

	return outVec_host;
}
